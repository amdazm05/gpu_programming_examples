
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <hip/hip_runtime_api.h>

//To profile this use : sudo nvprof --unified-memory-profiling off ./basic_kernel_example

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__
void initialise(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
  {
    y[i] = 2.0f;
	x[i] = 1.0f;
  }
}

__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}


int main(int argc, char **argv)
{
	
	int N = 1<<20;
	int blockSize = 512;
	int numBlocks = (N + blockSize - 1) / blockSize;

	float * x;
	float * y;
	
	gpuErrchk(hipMallocManaged(&x,N*sizeof(float)));
	gpuErrchk(hipMallocManaged(&y,N*sizeof(float)));

	std::cout<<"POST ALLOCATION"<<std::endl;
	
	initialise<<<numBlocks,blockSize>>>(N,x,y);
	add<<<numBlocks, blockSize>>>(N, x, y);


	hipFree(x);
	hipFree(y);

	hipDeviceReset();

	return EXIT_SUCCESS;
}

//some notes on profiling
//sudo nvprof --unified-memory-profiling off ./${executable} should make the profiling work 
//more here https://forums.developer.nvidia.com/t/nvprof-error-code-139-but-memcheck-ok/50329

